#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void cuda_hello() {
  printf("Hello World from GPU!\n");
}

int main() {
  cuda_hello<<<1,1>>>();
  hipDeviceSynchronize();
  return 0;
}
