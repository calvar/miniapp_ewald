#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdio>

__global__ void comp_kernel() {
  hipDoubleComplex z1 = make_hipDoubleComplex(3, -5);
  hipDoubleComplex z2 = make_hipDoubleComplex(1, 1);

  hipDoubleComplex p = make_hipDoubleComplex(3*cos(2.0), 3*sin(2.0));
  printf("%f,%f\n",hipCreal(p),hipCimag(p));
  
  double n = hipCreal( hipCmul(z1, hipConj(z1)) );
  printf("%f\n",n);

  z1 = hipCadd(z1, z2);
  printf("%f,%f\n",hipCreal(z1),hipCimag(z1));
}

int main() {
  comp_kernel<<<1,1>>>();
  hipDeviceSynchronize();
  
  return 0;
}
